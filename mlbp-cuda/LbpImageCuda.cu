#include "hip/hip_runtime.h"
#include "LbpImageCuda.h"
#include "hip/hip_runtime.h"
#include <iostream>
#include <stdexcept>
#include <cmath>
#include <cstdint>
#include <exception>
#include <string>

LbpImageCuda::LbpImageCuda(const std::vector<byte>& pixels, unsigned width, unsigned height) :
	AbstractLbpImage(pixels, width, height),
	d_pixels(nullptr),
	d_offsets(nullptr)
{
	// Move grayscale image to device
	hipMalloc((void**)&d_pixels, width * height);
	hipMemcpy((void*)d_pixels, (void*)this->pixels.data(), width * height, hipMemcpyHostToDevice);
	this->pixels.clear();
}

LbpImageCuda::~LbpImageCuda() {
	hipFree((void*)d_pixels);
}

__device__ bool isThreadInBounds(unsigned remainder) {
	// If this is the last block
	if(blockIdx.x + 1 == gridDim.x)
	{
		// If this thread lies over the array
		if(remainder > 0 && threadIdx.x >= remainder) {
			return false;
		}
	}
	return true;
}

__device__ long getGlobalIndex() {
	return threadIdx.x + blockIdx.x * blockDim.x;
}

__device__ float *getHistogram(float *histograms, unsigned row, unsigned col, unsigned histogramLength, unsigned blockEdge)
{
	int_pair block;
	block.y = row / blockEdge;
	block.x = col / blockEdge;
	long offset = histogramLength * (block.y * gridDim.x + block.x);
	return (histograms + offset);
}

__device__ byte pixelAt(byte *pixels, int row, unsigned width, int col)
{
	return pixels[row * width + col];
}

__device__ unsigned compareWithNeighborhood(byte *pixels, int_pair gaps_pixels, unsigned width, unsigned row, unsigned col, unsigned samples, int_pair *offsets)
{
	byte pixel = pixelAt(pixels, row, width, col);
	unsigned result = 0;

	for(int i = 0; i < samples; i++)
	{
		unsigned nrow = row + offsets[i].y;
		unsigned ncol = col + offsets[i].x;
		byte neighbor = pixelAt(pixels, nrow, width, ncol);

		if(pixel <= neighbor) {
			int shift = samples - 1 - i;
			result = result | (0x1 << shift);
		}
	}

	return result;
}

__global__ void writeZeroIntoHistograms(float *histograms, unsigned lastBlockRemainder)
{
	if(!isThreadInBounds(lastBlockRemainder)) {
		return;
	}
	histograms[getGlobalIndex()] = 0.f;
}

__global__ void computeLBPs(byte *pixels, int_pair gaps_pixels, unsigned width, float *histograms, unsigned histogramLength, unsigned numberHistograms, unsigned samples, unsigned blockEdge, int_pair *offsets)
{
	unsigned row = threadIdx.y + blockIdx.y * blockEdge;
	unsigned col = threadIdx.x + blockIdx.x * blockEdge;
	unsigned imageRow = row + gaps_pixels.y;
	unsigned imageCol = col + gaps_pixels.x;

	float *histogram = getHistogram(histograms, row, col, histogramLength, blockEdge);
	unsigned pattern = compareWithNeighborhood(pixels, gaps_pixels, width, imageRow, imageCol, samples, offsets);
	atomicAdd(&histogram[pattern], 1.f);
}

__global__ void normalizeHistograms(float *histograms, float divider, unsigned lastBlockRemainder)
{
	if(!isThreadInBounds(lastBlockRemainder)) {
		return;
	}
	histograms[getGlobalIndex()] /= divider;
}

float *LbpImageCuda::calculateNormalizedLBPs(float radius, unsigned samples, unsigned blockEdge)
{
	prepare(radius, samples, blockEdge);
	hipMalloc((void**)&d_offsets, sizeof(int_pair) * offsets.size());
	hipMemcpy((void*)d_offsets, (void*)offsets.data(), sizeof(int_pair) * offsets.size(), hipMemcpyHostToDevice);

	// Allocate device memory
	float *d_histograms;
	hipError_t error = hipMalloc((void**)&d_histograms, getHistogramsSizeInBytes());
	if(error) {
		throw std::invalid_argument("hipMalloc of " + std::to_string(getHistogramsSizeInBytes()) + " bytes has failed");
	}

	// Calculate CUDA grids
	dim3 lbpGridSize, lbpBlockSize;
	dim3 histGridSize, histBlockSize;
	unsigned remainder;
	calcLbpGridAndBlockSize(lbpGridSize, lbpBlockSize);
	calcHistGridAndBlockSize(histGridSize, histBlockSize, remainder);

	// Compute histrograms
	writeZeroIntoHistograms<<< histGridSize, histBlockSize >>>(
		d_histograms,
		remainder
	);
	computeLBPs<<< lbpGridSize, lbpBlockSize >>>(
		d_pixels,
		region.gaps_pixel,
		width,
		d_histograms,
		getHistogramLength(),
		getNumberHistograms(),
		samples,
		blockEdge,
		d_offsets
	);
	normalizeHistograms<<< histGridSize, histBlockSize >>>(
		d_histograms,
		blockEdge * blockEdge,
		remainder
	);

	// Copy result and return
	hipMemcpy(histograms, d_histograms, getHistogramsSizeInBytes(), hipMemcpyDeviceToHost);
	hipFree((void*)d_offsets);
	hipFree((void*)d_histograms);
	return histograms;
}

void LbpImageCuda::calcHistGridAndBlockSize(dim3& grid, dim3& block, unsigned& remainder)
{
	// Histograms will be covered by maximum-sized blocks
	auto props = getDeviceProps();
	auto numFloats = getHistogramsSizeInBytes() / sizeof(float);
	auto numBlocks = numFloats / props.maxThreadsPerBlock;
	remainder = numFloats % props.maxThreadsPerBlock;
	if(remainder > 0) {
		numBlocks++;
	}

	// Ensure max grid size is respected
	if(numBlocks > props.maxGridSize[0])
	{
		throw std::invalid_argument("maxGridSize is " + std::to_string(props.maxGridSize[0]) + " numBlocks is " + std::to_string(numBlocks));
	}
	grid = {(unsigned)numBlocks};
	block = {(unsigned)props.maxThreadsPerBlock};
}

void LbpImageCuda::calcLbpGridAndBlockSize(dim3& grid, dim3& block)
{
	// Ensure maximum sizes are respected
	auto props = getDeviceProps();
	if(blockEdge * blockEdge > props.maxThreadsPerBlock)
	{
		throw std::invalid_argument("Maximum block edge on this device is " + std::to_string((unsigned)std::sqrt(props.maxThreadsPerBlock)));
	}
	if(region.grid_size.x > props.maxGridSize[0] || region.grid_size.y > props.maxGridSize[1])
	{
		throw std::invalid_argument("Maximum grid size on this device is " + std::to_string(props.maxGridSize[0]) + "x" + std::to_string(props.maxGridSize[1]));
	}

	// Same as CPU implementation
	grid = {(unsigned)region.grid_size.x, (unsigned)region.grid_size.y};
	block = {blockEdge, blockEdge};
}

struct hipDeviceProp_t LbpImageCuda::getDeviceProps() {
	int device;
	hipGetDevice(&device);
	struct hipDeviceProp_t props;
	hipGetDeviceProperties(&props, device);
	return props;
}
