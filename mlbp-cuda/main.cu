#include <iostream>
#include <bitset>
#include <chrono>
#include "lodepng.h"
#include "LbpImageCpu.h"
#include "Benchmark.h"

#define BLOCK_EDGE 	16
#define RADIUS		1.0
#define SAMPLES		2
#define BLOCK_SIZE 	(BLOCK_EDGE*BLOCK_EDGE)

static std::vector<byte> pixels;
static unsigned width;
static unsigned height;

static bool loadImage(const std::string& filename)
{
	unsigned error = lodepng::decode(pixels, width, height, filename);
	if(error) {
		std::cerr << lodepng_error_text(error) << std::endl;
		return false;
	}
	std::cout << "image size is " << width << "x" << height << std::endl;
	return true;
}

int main(int argc, char **argv) {
	// Load
	if(argc < 2) {
		std::cerr << "No image file specified" << std::endl;
		return 1;
	}
	if(!loadImage(argv[1])) {
		return 1;
	}
	LbpImageCpu image(pixels, width, height);
	if(!image.checkMinimumSize(RADIUS, BLOCK_EDGE)) {
		std::cerr << "The image is too small" << std::endl;
		return 1;
	}
	// Prepare
	image.calcSamplingOffsets(RADIUS, SAMPLES);
	image.calcImageRegion(RADIUS, BLOCK_EDGE);

	// Process
	Benchmark::start();
	delete [] image.calculateNormalizedLBPs(RADIUS, BLOCK_EDGE, "test-output.png");
	Benchmark::stop();

	std::cout << Benchmark::getMillis() << "ms elapsed" << std::endl;
	return 0;
}
