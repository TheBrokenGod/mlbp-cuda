#include <iostream>
#include <bitset>
#include <chrono>
#include <stdexcept>
#include "lodepng.h"
#include "LbpImageCpu.h"
#include "LbpImageCuda.h"
#include "Benchmark.h"

static std::vector<byte> pixels;
static unsigned width;
static unsigned height;

static bool loadImage(const std::string& filename)
{
	unsigned error = lodepng::decode(pixels, width, height, filename);
	if(error) {
		std::cerr << lodepng_error_text(error) << std::endl;
		return false;
	}
	std::cout << "image size is " << width << "x" << height << std::endl;
	return true;
}

static void makeSampleOutput()
{
	LbpImageCpu output(pixels, width, height);
	delete output.calculateNormalizedLBPs(1, 8, 64, "test-output");

	float *histograms = output.calculateNormalizedLBPs(2, 4, 256);
	LbpImageCpu::saveHistogramsToFile(histograms, output.getHistogramLength(), output.getNumberHistograms(), "test-output");

	delete histograms;
	std::cerr << "Visual and textual output saved" << std::endl;
}


static void testAndBenchmark()
{
	LbpImageCpu image(pixels, width, height);
	LbpImageCuda d_Image(pixels, width, height);

	int samp[] = {4, 6, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20};
	float rads[] = {1.0, 1.75, 1.0, 1.0, 2.0, 2.75, 3.0, 3.0, 4.0, 4.0, 4.0, 5.0, 5.0, 5.0, 6.0};
	int edge[] = {16, 16, 16, 16, 32, 32, 32, 32, 32, 32, 32, 32, 32, 32, 32};
	for(int i = 0; i < 15; i++)
	{
		try {
			Benchmark::start();
			float *cpuHistograms = image.calculateNormalizedLBPs(rads[i], samp[i], edge[i]);
			Benchmark::stop();
			long cpuMillis = Benchmark::getMillis();

			Benchmark::start();
			float *gpuHistograms = d_Image.calculateNormalizedLBPs(rads[i], samp[i], edge[i]);
			Benchmark::stop();
			long gpuMillis = Benchmark::getMillis();

			std::cerr << "With conf {r=" << rads[i] << "; s=" << samp[i] << "; e=" << edge[i] << "} ";
			std::cerr << "\tCPU took " << cpuMillis << "ms and GPU " << gpuMillis << "ms";

			// Test against output correctness
			long limit = image.getNumberHistograms() * image.getHistogramLength();
			for(long j = 0; j < limit; j++)
			{
				if(cpuHistograms[j] != gpuHistograms[j]) {
					throw std::logic_error("CPU and GPU outputs differ at " + std::to_string(j) + " " + std::to_string(cpuHistograms[j]) + " " + std::to_string(gpuHistograms[j]));
				}
			}
			delete cpuHistograms, gpuHistograms;
			std::cerr << "\tTest finished OK" << std::endl;
		}
		catch(const std::bad_alloc& e) {
			std::cerr << "Conf {r=" << rads[i] << "; s=" << samp[i] << "; e=" << edge[i] << "} is not supported" << std::endl;
			return;
		}
		catch(const std::invalid_argument& e) {
			std::cerr << e.what() << std::endl;
			std::cerr << "Conf {r=" << rads[i] << "; s=" << samp[i] << "; e=" << edge[i] << "} is not supported" << std::endl;
			return;
		}
	}
}

int main(int argc, char **argv) {
	if(argc < 2) {
		std::cerr << "No image file specified" << std::endl;
		return 1;
	}
	if(!loadImage(argv[1])) {
		return 1;
	}
	makeSampleOutput();
	testAndBenchmark();
	hipDeviceReset();
	return 0;
}
