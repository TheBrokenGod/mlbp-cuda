#include <iostream>
#include <bitset>
#include <chrono>
#include <stdexcept>
#include "lodepng.h"
#include "LbpImageCpu.h"
#include "LbpImageCuda.h"
#include "Benchmark.h"

static std::vector<byte> pixels;
static unsigned width;
static unsigned height;

static bool loadImage(const std::string& filename)
{
	unsigned error = lodepng::decode(pixels, width, height, filename);
	if(error) {
		std::cerr << lodepng_error_text(error) << std::endl;
		return false;
	}
	std::cout << "image size is " << width << "x" << height << std::endl;
	return true;
}

static void makeSampleOutput()
{
	LbpImageCpu output(pixels, width, height);
	output.calculateNormalizedLBPs(5, 8, 64, "test-output");

	auto histograms = output.calculateNormalizedLBPs(2, 4, 256);
	LbpImageCpu::saveHistogramsToFile(histograms, output.getHistogramLength(), output.getNumberHistograms(), "test-output");

	std::cerr << "Visual and textual output saved" << std::endl;
}


static void testAndBenchmark()
{
	LbpImageCpu image(pixels, width, height);
	LbpImageCuda d_Image(pixels, width, height);

	int samp[] = {4, 6, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21};
	float rads[] = {1.0, 1.75, 1.0, 1.0, 2.0, 2.75, 3.0, 3.0, 4.0, 4.0, 4.0, 5.0, 5.0, 5.0, 6.0, 6.0};
	int edge[] = {16, 16, 16, 16, 32, 32, 32, 32, 32, 32, 32, 32, 32, 32, 32, 32};
	for(int i = 0; i < 16; i++)
	{
		try {
			Benchmark::start();
			auto cpuHistograms = image.calculateNormalizedLBPs(rads[i], samp[i], edge[i]);
			Benchmark::stop();
			long cpuMillis = Benchmark::getMillis();

			Benchmark::start();
			auto gpuHistograms = d_Image.calculateNormalizedLBPs(rads[i], samp[i], edge[i]);
			Benchmark::stop();
			long gpuMillis = Benchmark::getMillis();

			std::cerr << "With conf {s=" << samp[i] << "; r=" << rads[i] << "; e=" << edge[i] << "} ";
			std::cerr << "\tCPU took " << cpuMillis << "ms and GPU " << gpuMillis << "ms";

			// Test against output correctness
			long limit = image.getNumberHistograms() * image.getHistogramLength();
			for(long j = 0; j < limit; j++)
			{
				if(cpuHistograms[j] != gpuHistograms[j]) {
					throw std::logic_error("CPU and GPU outputs differ at " + std::to_string(j) + " " + std::to_string(cpuHistograms[j]) + " " + std::to_string(gpuHistograms[j]));
				}
			}
			std::cerr << "\tTest finished OK" << std::endl;
		}
		catch(const std::invalid_argument& e) {
			std::cerr << e.what() << std::endl;
			std::cerr << "Conf {s=" << samp[i] << "; r=" << rads[i] << "; e=" << edge[i] << "} is not supported" << std::endl;
			std::cerr << "End of main" << std::endl;
			return;
		}
	}
}

int main(int argc, char **argv) {
	if(argc < 2) {
		std::cerr << "No image file specified" << std::endl;
		return 1;
	}
	if(!loadImage(argv[1])) {
		return 1;
	}
	makeSampleOutput();
	testAndBenchmark();
	hipDeviceReset();
	return 0;
}
